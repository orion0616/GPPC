#include "hip/hip_runtime.h"
#include <deque>
#include <queue>
#include <unordered_map>
#include <vector>
#include <algorithm>
#include <assert.h>
#include <iostream>
#include <cmath>
#include "Entry.h"
#include "state.h"
#include "BinaryHeap.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#define N 512
#define  RAW(x) thrust::raw_pointer_cast(x)

std::vector<bool> map;
std::vector<int> visited;
std::vector<xyLoc> succ;
int width, height;

const char *GetName()
{
    return "TestProgram";
}

void PreprocessMap(std::vector<bool> &bits, int width, int height, const char *filename)
{
    printf("Not writing to file '%s'\n", filename);
}

void *PrepareForSearch(std::vector<bool> &bits, int w, int h, const char *filename)
{
    printf("Not reading from file '%s'\n", filename);
    map = bits;
    width = w;
    height = h;
    return (void *)13182;
}

__device__ state goal;
__device__ state candidate;

__global__ void remove(state*** arrays, int* lengths, state** S, int* lengthForS) {
    int num = threadIdx.x;
    state* array = array[num];
    int len = lengths[num];

    BinaryHeap pq = BinaryHeap(array, len);
    if (pq.empty()) {
        return;
    }
    state* min = pq.remove();
    if (min->node.x == goal.node.x && min->node.y == goal.node.y) {
        if (candidate.isNil() || min.f < candidate.f) {
            candidate = min;
        }
    }

    lengthForS[num] = GetSuccessors_for_gastar(min, S[num]);
    return;
}

bool isAllQueueEmpty(int* lengths) {
    for(int i=0; i< N; i++){
        if(lengths[i] != 0)
            return false;
    }
    return true;
}

__global__ void duplicate_detection(state** table, state* s, state*** pqs, int* lengths, int rand) {
    state old = state[s->hash()];
    if (!old->isNil() && old->g_value < s->g_value) {
        return;
    } else {
        /* s->f_value = s->f_value */
        hiprandState_t state;
        hiprand_init(0,0,0, &state);
        int result = hiprand(&state) %N;
        BinaryHeap pq = BinaryHeap(pqs[result], lengths[result]);
        pq.add(s);
        lengths[num] += 1;
    }
}

bool GetPath_GASTAR(void *data, xyLoc s, xyLoc g, std::vector<xyLoc> &path) {
    assert((long)data == 13182);

    thrust::device_vector<thrust::device_vector<state> > pqs;
    for(int i=0;i<N;i++){
        thrust::device_vector<state> tmp;
        pqs.push_back(tmp);
    }
    thrust::device_vector<int> lengths(N);
    lengths[0] = 1;
    pqs[0].push_back(state);
    thrust::device_vector<&state> table;

    if(path.size() > 0) {
        path.push_back(g);
        return true;
    }

    while(!isAllQueueEmpty(lengths)) {
        thrust::device_vector<thrust::device_vector<state> > S;
        for(int i=0;i<N;i++){
            thrust::device_vector<state> succ(8);
            pqs.push_back(succ);
        }
        thrust::device_vector<state> lengthForS(N,0);
        // XXX:parallel
        remove<<<1, N>>>(RAW(pqs), RAW(lengths), RAW(S), RAW(lengthForS));
        bool flag = true;
        if (!candidate.isNil()) {
            for(int i=0; i< N; i++) {
                if (pqs[i].empty()){
                    continue;
                } else {
                    if(candidate.f_value > pqs[i].top()->f_value)
                        flag = false;
                    break;
                }
            }
        }
        if(flag){
            break;
        }
        // XXX:parallel
        duplicate_detection<<<1,N>>>(RAW(table), RAW(s), RAW(pqs));
    }

    // TODO: must change
    while(last.x != s.x || last.y != s.y) {
        xyLoc pos;
        pos.x = last.x;
        pos.y = last.y;
        path.push_back(pos);
        last = *last.parent;
    }
    path.push_back(s);

    if(path.size() > 0) {
        path.pop_back();
        return false;
    }
    return true;
}

int GetIndex(xyLoc s)
{
    return s.y*width+s.x;
}

int GetSuccessors_for_gastar(state &s, state* neighbots) {
    return 0;
}

/* void GetSuccessors_for_astar(node &s, std::vector<node> &neighbors, xyLoc g) {     */
/*     bool up = false, down = false, left = false, right = false;                    */
/*     neighbors.resize(0);                                                           */

/*     node next = create_next_node(s,g,RIGHT);                                       */
/*     if (next.x < width && map[GetIndex(next)]) {                                   */
/*         neighbors.push_back(next);                                                 */
/*         right = true;                                                              */
/*     }                                                                              */
/*     next = create_next_node(s,g,LEFT);                                             */
/*     if (next.x >= 0 && map[GetIndex(next)]) {                                      */
/*         neighbors.push_back(next);                                                 */
/*         left = true;                                                               */
/*     }                                                                              */
/*     next = create_next_node(s,g,UP);                                               */
/*     if (next.y >= 0 && map[GetIndex(next)]) {                                      */
/*         neighbors.push_back(next);                                                 */
/*         up = true;                                                                 */
/*     }                                                                              */
/*     next = create_next_node(s,g,DOWN);                                             */
/*     if (next.y < height && map[GetIndex(next)]) {                                  */
/*         neighbors.push_back(next);                                                 */
/*         down = true;                                                               */
/*     }                                                                              */
/*     next = create_next_node(s,g,LOWERRIGHT);                                       */
/*     if (next.y < height && next.x < width && map[GetIndex(next)] && right && down) */
/*         neighbors.push_back(next);                                                 */
/*     next = create_next_node(s,g,UPPERRIGHT);                                       */
/*     if (next.y >= 0 &&  next.x < width && map[GetIndex(next)] && right && up)      */
/*         neighbors.push_back(next);                                                 */
/*     next = create_next_node(s,g,UPPERLEFT);                                        */
/*     if (next.y >= 0 && next.x >= 0 && map[GetIndex(next)] && left && up)           */
/*         neighbors.push_back(next);                                                 */
/*     next = create_next_node(s,g,LOWERLEFT);                                        */
/*     if (next.y < height && next.x >= 0 && map[GetIndex(next)] && left && down)     */
/*         neighbors.push_back(next);                                                 */
/* }                                                                                  */

// make plan
// trace path by visited
void ExtractPath(xyLoc end, std::vector<xyLoc> &finalPath)
{
    int currCost = visited[GetIndex(end)];

    finalPath.resize(0);
    finalPath.push_back(end);

    while (currCost != 1)
    {
        GetSuccessors(finalPath.back(), succ);
        for (unsigned int x = 0; x < succ.size(); x++)
        {
            if (visited[GetIndex(succ[x])] == currCost-1)
            {
                finalPath.push_back(succ[x]);
                currCost--;
                break;
            }
        }
    }
    std::reverse(finalPath.begin(), finalPath.end());
}

